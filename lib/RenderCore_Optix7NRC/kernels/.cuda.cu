#include "hip/hip_runtime.h"
/* .cuda.cu - Copyright 2019/2020 Utrecht University

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

	   http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/

#include ".cuda.h"

namespace lh2core
{

// path tracing buffers and global variables
__constant__ CoreInstanceDesc* instanceDescriptors;
__constant__ CUDAMaterial* materials;
__constant__ CoreLightTri* triLights;
__constant__ CorePointLight* pointLights;
__constant__ CoreSpotLight* spotLights;
__constant__ CoreDirectionalLight* directionalLights;
__constant__ int4 lightCounts;			// area, point, spot, directional
__constant__ uchar4* argb32;
__constant__ float4* argb128;
__constant__ uchar4* nrm32;
__constant__ float4* skyPixels;
__constant__ int skywidth;
__constant__ int skyheight;
__constant__ PathState* pathStates;
__constant__ float4* debugData;
__constant__ LightCluster* lightTree;

__constant__ mat4 worldToSky;

// path tracer settings
__constant__ __device__ float geometryEpsilon;
__constant__ __device__ float clampValue;

// staging: copies will be batched and carried out after rendering completes, 
// to allow the CPU to update the scene concurrently with GPU rendering.

enum { INSTS = 0, MATS, TLGHTS, PLGHTS, SLGHTS, DLGHTS, LCNTS, RGB32, RGBH, NRMLS, SKYPIX, SKYW, SKYH, SMAT, DBGDAT, GEPS, CLMPV, LTREE };

// device pointers are not real pointers for nvcc, so we need a bit of a hack.

struct StagedPtr { void* p; int id; };
struct StagedInt { int v; int id; };
struct StagedInt4 { int4 v; int id; };
struct StagedFloat3 { float3 v; int id; };
struct StagedMat { mat4 v; int id; };
struct StagedF32 { float v; int id; };
struct StagedCpy { void* d; void* s; int n; };
static std::vector<StagedPtr> stagedPtr;
static std::vector<StagedInt> stagedInt;
static std::vector<StagedInt4> stagedInt4;
static std::vector<StagedFloat3> stagedFloat3;
static std::vector<StagedMat> stagedMat;
static std::vector<StagedF32> stagedF32;
static std::vector<StagedCpy> stagedCpy;

__host__ static void pushPtrCpy( int id, void* p )
{
	if (id == INSTS) hipMemcpyToSymbol(HIP_SYMBOL( instanceDescriptors), &p, sizeof( void* ) );
	if (id == MATS) hipMemcpyToSymbol(HIP_SYMBOL( materials), &p, sizeof( void* ) );
	if (id == TLGHTS) hipMemcpyToSymbol(HIP_SYMBOL( triLights), &p, sizeof( void* ) );
	if (id == PLGHTS) hipMemcpyToSymbol(HIP_SYMBOL( pointLights), &p, sizeof( void* ) );
	if (id == SLGHTS) hipMemcpyToSymbol(HIP_SYMBOL( spotLights), &p, sizeof( void* ) );
	if (id == DLGHTS) hipMemcpyToSymbol(HIP_SYMBOL( directionalLights), &p, sizeof( void* ) );
	if (id == RGB32) hipMemcpyToSymbol(HIP_SYMBOL( argb32), &p, sizeof( void* ) );
	if (id == RGBH) hipMemcpyToSymbol(HIP_SYMBOL( argb128), &p, sizeof( void* ) );
	if (id == NRMLS) hipMemcpyToSymbol(HIP_SYMBOL( nrm32), &p, sizeof( void* ) );
	if (id == SKYPIX) hipMemcpyToSymbol(HIP_SYMBOL( skyPixels), &p, sizeof( void* ) );
	if (id == DBGDAT) hipMemcpyToSymbol(HIP_SYMBOL( debugData), &p, sizeof( void* ) );
	if (id == LTREE) hipMemcpyToSymbol(HIP_SYMBOL( lightTree), &p, sizeof( void* ) );
}
__host__ static void pushIntCpy( int id, const int v )
{
	if (id == SKYW) hipMemcpyToSymbol(HIP_SYMBOL( skywidth), &v, sizeof( int ) );
	if (id == SKYH) hipMemcpyToSymbol(HIP_SYMBOL( skyheight), &v, sizeof( int ) );
}
__host__ static void pushF32Cpy( int id, const float v )
{
	if (id == GEPS) hipMemcpyToSymbol(HIP_SYMBOL( geometryEpsilon), &v, sizeof( float ) );
	if (id == CLMPV) hipMemcpyToSymbol(HIP_SYMBOL( clampValue), &v, sizeof( int ) );
}
__host__ static void pushMatCpy( int id, const mat4& m )
{
	if (id == SMAT) hipMemcpyToSymbol(HIP_SYMBOL( worldToSky), &m, sizeof( mat4 ) );
}
__host__ static void pushInt4Cpy( int id, const int4& v )
{
	if (id == LCNTS) hipMemcpyToSymbol(HIP_SYMBOL( lightCounts), &v, sizeof( int4 ) );
}
__host__ static void pushFloat3Cpy( int id, const float3& v )
{
	// nothing here yet
}

#define MAXVARS	32
static void* prevPtr[MAXVARS] = {};
static int prevInt[MAXVARS] = {};
static float prevFloat[MAXVARS] = {};
static int4 prevInt4[MAXVARS] = {};
// static float3 prevFloat3[MAXVARS] = {};
static bool prevValSet[MAXVARS] = {};

__host__ static void stagePtrCpy( int id, void* p )
{
	if (prevPtr[id] == p) return; // not changed
	StagedPtr n = { p, id };
	stagedPtr.push_back( n );
	prevPtr[id] = p;
}
__host__ static void stageIntCpy( int id, const int v )
{
	if (prevValSet[id] == true && prevInt[id] == v) return;
	StagedInt n = { v, id };
	stagedInt.push_back( n );
	prevValSet[id] = true;
	prevInt[id] = v;
}
__host__ static void stageF32Cpy( int id, const float v )
{
	if (prevValSet[id] == true && prevFloat[id] == v) return;
	StagedF32 n = { v, id };
	stagedF32.push_back( n );
	prevValSet[id] = true;
	prevFloat[id] = v;
}
__host__ static void stageMatCpy( int id, const mat4& m ) { StagedMat n = { m, id }; stagedMat.push_back( n ); }
__host__ static void stageInt4Cpy( int id, const int4& v )
{
	if (prevValSet[id] == true && prevInt4[id].x == v.x && prevInt4[id].y == v.y && prevInt4[id].z == v.z && prevInt4[id].w == v.w) return;
	StagedInt4 n = { v, id };
	stagedInt4.push_back( n );
	prevValSet[id] = true;
	prevInt4[id] = v;
}
/* __host__ static void stageFloat3Cpy( int id, const float3& v )
{
	if (prevValSet[id] == true && prevFloat3[id].x == v.x && prevFloat3[id].y == v.y && prevFloat3[id].z == v.z) return;
	StagedFloat3 n = { v, id };
	stagedFloat3.push_back( n );
	prevValSet[id] = true;
	prevFloat3[id] = v;
} */

__host__ void stageMemcpy( void* d, void* s, int n ) { StagedCpy c = { d, s, n }; stagedCpy.push_back( c ); }

__host__ void stageInstanceDescriptors( CoreInstanceDesc* p ) { stagePtrCpy( INSTS /* instanceDescriptors */, p ); }
__host__ void stageMaterialList( CUDAMaterial* p ) { stagePtrCpy( MATS /* materials */, p ); }
__host__ void stageTriLights( CoreLightTri* p ) { stagePtrCpy( TLGHTS /* triLights */, p ); }
__host__ void stagePointLights( CorePointLight* p ) { stagePtrCpy( PLGHTS /* pointLights */, p ); }
__host__ void stageSpotLights( CoreSpotLight* p ) { stagePtrCpy( SLGHTS /* spotLights */, p ); }
__host__ void stageDirectionalLights( CoreDirectionalLight* p ) { stagePtrCpy( DLGHTS /* directionalLights */, p ); }
__host__ void stageARGB32Pixels( uint* p ) { stagePtrCpy( RGB32 /* argb32 */, p ); }
__host__ void stageARGB128Pixels( float4* p ) { stagePtrCpy( RGBH /* argb128 */, p ); }
__host__ void stageNRM32Pixels( uint* p ) { stagePtrCpy( NRMLS /* nrm32 */, p ); }
__host__ void stageSkyPixels( float4* p ) { stagePtrCpy( SKYPIX /* skyPixels */, p ); }
__host__ void stageSkySize( int w, int h ) { stageIntCpy( SKYW /* skywidth */, w ); stageIntCpy( SKYH /* skyheight */, h ); }
__host__ void stageWorldToSky( const mat4& worldToLight ) { stageMatCpy( SMAT /* worldToSky */, worldToLight ); }
__host__ void stageDebugData( float4* p ) { stagePtrCpy( DBGDAT /* debugData */, p ); }
__host__ void stageGeometryEpsilon( float e ) { stageF32Cpy( GEPS /* geometryEpsilon */, e ); }
__host__ void stageClampValue( float c ) { stageF32Cpy( CLMPV /* clampValue */, c ); }
__host__ void stageLightTree( LightCluster* t ) { stagePtrCpy( LTREE /* light tree */, t ); }
__host__ void stageLightCounts( int tri, int point, int spot, int directional )
{
	const int4 counts = make_int4( tri, point, spot, directional );
	stageInt4Cpy( LCNTS /* lightCounts */, counts );
}

__host__ void pushStagedCopies()
{
	for (auto c : stagedCpy) hipMemcpy( c.d, c.s, c.n, hipMemcpyHostToDevice ); stagedCpy.clear();
	for (auto n : stagedPtr) pushPtrCpy( n.id, n.p ); stagedPtr.clear();
	for (auto n : stagedInt) pushIntCpy( n.id, n.v ); stagedInt.clear();
	for (auto n : stagedInt4) pushInt4Cpy( n.id, n.v ); stagedInt4.clear();
	for (auto n : stagedFloat3) pushFloat3Cpy( n.id, n.v ); stagedFloat3.clear();
	for (auto n : stagedF32) pushF32Cpy( n.id, n.v ); stagedF32.clear();
	for (auto n : stagedMat) pushMatCpy( n.id, n.v ); stagedMat.clear();
}

// counters for persistent threads
static __device__ Counters* counters;
__global__ void InitCountersForExtend_Kernel( int pathCount )
{
	if (threadIdx.x != 0) return;
	counters->activePaths = pathCount;	// remaining active paths
	counters->shaded = 0;				// persistent thread atomic for shade kernel
	counters->generated = 0;			// persistent thread atomic for generate in .optix.cu
	counters->extensionRays = 0;		// compaction counter for extension rays
	counters->shadowRays = 0;			// compaction counter for connections
	counters->connected = 0;
	counters->totalExtensionRays = pathCount;
	counters->totalShadowRays = 0;
}
__host__ void InitCountersForExtend( int pathCount ) { InitCountersForExtend_Kernel << <1, 32 >> > (pathCount); }
__global__ void InitCountersSubsequent_Kernel()
{
	if (threadIdx.x != 0) return;
	counters->totalExtensionRays += counters->extensionRays;
	counters->activePaths = counters->extensionRays;	// remaining active paths
	counters->extended = 0;				// persistent thread atomic for genSecond in .optix.cu
	counters->shaded = 0;				// persistent thread atomic for shade kernel
	counters->extensionRays = 0;		// compaction counter for extension rays
}
__host__ void InitCountersSubsequent() { InitCountersSubsequent_Kernel << <1, 32 >> > (); }
__host__ void SetCounters( Counters* p ) { hipMemcpyToSymbol(HIP_SYMBOL( counters), &p, sizeof( void* ) ); }

// nrc auxiliary counters
static __device__ NRCCounters* nrcCounters;
__host__ void SetNRCCounters( NRCCounters* p ) { hipMemcpyToSymbol(HIP_SYMBOL(nrcCounters), &p, sizeof(void*)); }

// functional blocks
#include "tools_shared.h"
#include "sampling_shared.h"
#include "material_shared.h"
#include "lights_shared.h"
#include "bsdf.h"
#include "pathtracer.h"
#include "finalize_shared.h"

#if __CUDA_ARCH__ > 700 // Volta deliberately excluded
__global__  __launch_bounds__(128 /* max block size */, 2 /* min blocks per sm TURING */)
#else
__global__  __launch_bounds__(256 /* max block size */, 2 /* min blocks per sm, PASCAL, VOLTA */)
#endif
__global__ void PrepareNRCTrainData_Kernel( float4* trainBuf, float4* trainInputBuf, float4* debugView ) {
	int jobIndex = threadIdx.x + blockIdx.x * blockDim.x;
	if (jobIndex >= NRC_NUMTRAINRAYS) {
		return;
	}

	float3 luminances[NRC_MAXTRAINPATHLENGTH];
	bool previousDataValid = false;
	uint lastValidPathLength = 0;

	for (uint pathLength = NRC_MAXTRAINPATHLENGTH; pathLength >= 1; pathLength--) {
		const float4 data0 = trainBuf[(NRC_MAXTRAINPATHLENGTH * NRC_TRAINCOMPONENTSIZE) * jobIndex + (pathLength - 1) * NRC_TRAINCOMPONENTSIZE + 0];
		const float4 data1 = trainBuf[(NRC_MAXTRAINPATHLENGTH * NRC_TRAINCOMPONENTSIZE) * jobIndex + (pathLength - 1) * NRC_TRAINCOMPONENTSIZE + 1];
		const float4 data2 = trainBuf[(NRC_MAXTRAINPATHLENGTH * NRC_TRAINCOMPONENTSIZE) * jobIndex + (pathLength - 1) * NRC_TRAINCOMPONENTSIZE + 2];
		const float4 data3 = trainBuf[(NRC_MAXTRAINPATHLENGTH * NRC_TRAINCOMPONENTSIZE) * jobIndex + (pathLength - 1) * NRC_TRAINCOMPONENTSIZE + 3];
		const float4 data4 = trainBuf[(NRC_MAXTRAINPATHLENGTH * NRC_TRAINCOMPONENTSIZE) * jobIndex + (pathLength - 1) * NRC_TRAINCOMPONENTSIZE + 4];
		const float4 data5 = trainBuf[(NRC_MAXTRAINPATHLENGTH * NRC_TRAINCOMPONENTSIZE) * jobIndex + (pathLength - 1) * NRC_TRAINCOMPONENTSIZE + 5];

		const uint flags = __float_as_uint(data4.w);

		if ((flags & S_NRC_DATA_VALID) > 0 && !previousDataValid) {
			// This is the last bounce, reason:
			// 1. NRC_MAXTRAINPATHLENGTH exceed, or killed by russian roulette
			//   (in this case no luminances other than direct lighting will occur)
			// 2. hit emissive material
			// 3. hit skybox

			// NOTE: possible sources
			// 1. emissive
			// 2. hit skybox
			// 3. direct lighting from one of the lights
			float3 directLuminance = make_float3(data4.x, data4.y, data4.z);
			
			luminances[pathLength - 1] = directLuminance;
			previousDataValid = true;
			lastValidPathLength = pathLength;
		}
		else if ((flags & S_NRC_DATA_VALID) > 0 && previousDataValid) {
			// NOTE: only direct lighting from one of the lights are possible here
			float3 directLuminance = make_float3(data4.x, data4.y, data4.z);

			float3 segmentThroughput = make_float3(data5.x, data5.y, data5.z);
			float3 indirectLuminance = segmentThroughput * luminances[pathLength];
			luminances[pathLength - 1] = directLuminance + indirectLuminance;
		}
		else if ((flags & S_NRC_DATA_VALID) == 0 && previousDataValid) {
			// illegal data encountered, TODO: error recovery
			NRC_DUMP_WARN("[WARN] illegal data, jobIndex=%d, current pathLength=%d", jobIndex, pathLength);
			return;
		}
	}
	// TODO: debugView

	if (!previousDataValid) {
		// TODO: error recovery
		NRC_DUMP_WARN("[WARN] no valid data, jobIndex=%d", jobIndex);
		return;
	}

	for (uint pathLength = lastValidPathLength; pathLength >= 1; pathLength--) {
		const uint raySegmentIdx = atomicAdd( &nrcCounters->nrcActualTrainRays, 1);

		const float4 data0 = trainBuf[(NRC_MAXTRAINPATHLENGTH * NRC_TRAINCOMPONENTSIZE) * jobIndex + (pathLength - 1) * NRC_TRAINCOMPONENTSIZE + 0];
		const float4 data1 = trainBuf[(NRC_MAXTRAINPATHLENGTH * NRC_TRAINCOMPONENTSIZE) * jobIndex + (pathLength - 1) * NRC_TRAINCOMPONENTSIZE + 1];
		const float4 data2 = trainBuf[(NRC_MAXTRAINPATHLENGTH * NRC_TRAINCOMPONENTSIZE) * jobIndex + (pathLength - 1) * NRC_TRAINCOMPONENTSIZE + 2];
		const float4 data3 = trainBuf[(NRC_MAXTRAINPATHLENGTH * NRC_TRAINCOMPONENTSIZE) * jobIndex + (pathLength - 1) * NRC_TRAINCOMPONENTSIZE + 3];
		const float4 data4 = trainBuf[(NRC_MAXTRAINPATHLENGTH * NRC_TRAINCOMPONENTSIZE) * jobIndex + (pathLength - 1) * NRC_TRAINCOMPONENTSIZE + 4];
		const float4 data5 = trainBuf[(NRC_MAXTRAINPATHLENGTH * NRC_TRAINCOMPONENTSIZE) * jobIndex + (pathLength - 1) * NRC_TRAINCOMPONENTSIZE + 5];

		// TODO: ray origin => ray intersection point
		// float3 intersection
	}
}

__host__ void PrepareNRCTrainData() {}

} // namespace lh2core

// EOF